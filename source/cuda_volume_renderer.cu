#include "hip/hip_runtime.h"
#ifndef CUDA_VOLUME_RENDERER_CU
#define CUDA_VOLUME_RENDERER_CU

//#include "Common.h"
#ifdef WIN32
//#include <GL/glew.h>
//#include <GLFW/glfw3.h>
#else

//#include <GLES3/gl3.h>
//#include <GLES3/gl3ext.h>
#endif
#include "../3rdparty/cudaHelper/hip/hip_runtime_api.h"
#include "../3rdparty/cudaHelper/hip/hip_vector_types.h"
//#include "../3rdparty/cudaHelper/helper_cuda_gl.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
//#include <cuda_gl_interop.h>



typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;

typedef unsigned char VolumeType;
//typedef unsigned short VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture

typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};


__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__global__ void
d_render(uint *d_output, uint imageW, uint imageH, float3x4 invViewMatrix, float aspectRatio, float maxRaySteps, float rayStepSize)
{
    //const int maxSteps = 500;
    //const float tstep = 0.01f;
    const float opacityThreshold = 1.0f;
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;

    float u = ((x / (float) imageW)*2.0f-1.0f) * aspectRatio;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in model space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(invViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step = eyeRay.d*rayStepSize;

    for (int i=0; i<maxRaySteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1] coordinates
		//float xx =pos.x*0.5f+0.5f;
		//float yy =pos.y*0.5f+0.5f;
		//float zz =pos.z*0.5f+0.5f;
		//float sample = 0;
        float sample = tex3D(tex, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
        //sample *= 64.0f;    // scale for 10-bit data

        // lookup in transfer function texture
        //float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);
        float4 col = tex1D(transferTex, sample);
		//float4 col = make_float4(0);
       // col.w *= 0.05f;

        // "under" operator for back-to-front blending
        //sum = lerp(sum, col, col.w);

        // pre-multiply alpha
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;
        // "over" operator for front-to-back blending
        sum = sum + col*(1.0f - sum.w);

        // exit early if opaque
        if (t > tfar || sum.w > opacityThreshold)
		{
			//sum += make_float4(1.0f) * (1.0f - sum.w);
            break;
		}

        t += rayStepSize;
        pos += step;
    }

   // sum *= brightness;

    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(sum);
}



extern "C"
void initCuda()
{
//	    // Otherwise pick the device with highest Gflops/s
//    int devID = gpuGetMaxGflopsDeviceId();
//    cudaGLSetGLDevice(devID);
}
extern "C" void exitCuda()
{
    checkCudaErrorsLog(hipFreeArray(d_volumeArray));
    checkCudaErrorsLog(hipFreeArray(d_transferFuncArray));
}

extern "C" void writeVolume(void *volume, hipExtent volumeSize)
{
	if(d_volumeArray)
	{
		    checkCudaErrorsLog(hipFreeArray(d_volumeArray));
	}
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrorsLog(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrorsLog(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    tex.addressMode[1] = hipAddressModeClamp;

    // bind array to 3D texture
    checkCudaErrorsLog(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
}

extern "C" void writeTransferFunction(float *transferFunction, int transferFunctionSize)
{
	if(d_transferFuncArray)
	{
		checkCudaErrorsLog(hipFreeArray(d_transferFuncArray));
	}
	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;
    checkCudaErrorsLog(hipMallocArray(&d_transferFuncArray, &channelDesc2, transferFunctionSize, 1));
    checkCudaErrorsLog(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunction, transferFunctionSize * 4 * sizeof(float), hipMemcpyHostToDevice));
    //checkCudaErrorsLog(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
    //checkCudaErrorsLog(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));


    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    checkCudaErrorsLog(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));

}

extern "C" void initCudaVolume(void *volume, hipExtent volumeSize, float *transferFunction, int transferFunctionSize)
{
	    // create 3D array
	writeVolume(volume, volumeSize);
	writeTransferFunction(transferFunction, transferFunctionSize);

    //float4 transferFunc[] =
    //{
    //    {  0.0, 0.0, 0.0, 0.0, },
    //    {  1.0, 0.0, 0.0, 1.0, },
    //    {  1.0, 0.5, 0.0, 1.0, },
    //    {  1.0, 1.0, 0.0, 1.0, },
    //    {  0.0, 1.0, 0.0, 1.0, },
    //    {  0.0, 1.0, 1.0, 1.0, },
    //    {  0.0, 0.0, 1.0, 1.0, },
    //    {  1.0, 0.0, 1.0, 1.0, },
    //    {  0.0, 0.0, 0.0, 0.0, },
    //};
	//CT-Knee transfer function
    //float4 transferFunc[] =
    //{
    //    {  0.0, 0.0, 0.0, 0.0, },
    //    {  1.0, 0.0, 0.0, 8.0f/100.0f, },
    //    {  1.0, 95.0f/255.0f, 0.0, 0.0, },
    //    {  1.0, 191.0f/255.0f, 0.0, 0.0, },
    //    {  223.0f/255.0f, 1.0f, 0.0, 1.0f/100.0f, },
    //    {  127.0f/255.0f, 1.0f, 0.0, 12.0f/100.0f, },
    //    {  31.0f/255.0f, 1.0f, 0.0, 2.0f/100.0f, },
    //    {  0.0, 1.0f, 63.0f/255.0f, 5.0/100.0f, },
    //    {  0.0, 1.0f, 159.0f/255.0f, 24.0f/100.0f, },
    //    {  0.0, 1.0f, 1.0f, 18.0f/100.0f, },
    //    {  0.0, 159.0f/255.0f, 1.0f, 24.0f/100.0f, },
    //    {  0.0, 63.0f/255.0f, 1.0f, 30.0f/100.0f, },
    //    {  31.0f/255.0f, 0.0, 1.0f, 34.0f/100.0f, },
    //    {  127.0f/255.0f, 0.0, 1.0f, 24.0f/100.0f, },
    //    {  223.0f/255.0f, 0.0, 1.0f, 21.0f/100.0f, },
    //    {  1.0f, 0.0, 191.0f/255.0f, 53.0f/100.0f, },
    //    {  1.0f, 0.0, 95.0f/255.0f, 61.0f/100.0f, },
    //    {  1.0f, 1.0f, 1.0f, 0.0f/100.0f, },
    //};
}


extern "C" void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH, float *invViewMatrix, float aspectRatio, float maxRaySteps, float rayStepSize)
{
	float3x4 cudaInvViewMatrix;
	cudaInvViewMatrix.m[0].x = invViewMatrix[0];
	cudaInvViewMatrix.m[0].y = invViewMatrix[1];
	cudaInvViewMatrix.m[0].z = invViewMatrix[2];
	cudaInvViewMatrix.m[0].w = invViewMatrix[3];

	cudaInvViewMatrix.m[1].x = invViewMatrix[4];
	cudaInvViewMatrix.m[1].y = invViewMatrix[5];
	cudaInvViewMatrix.m[1].z = invViewMatrix[6];
	cudaInvViewMatrix.m[1].w = invViewMatrix[7];

	cudaInvViewMatrix.m[2].x = invViewMatrix[8];
	cudaInvViewMatrix.m[2].y = invViewMatrix[9];
	cudaInvViewMatrix.m[2].z = invViewMatrix[10];
	cudaInvViewMatrix.m[2].w = invViewMatrix[11];
    d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, cudaInvViewMatrix, aspectRatio, maxRaySteps, rayStepSize);
}


extern "C" void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrorsLog(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}




#endif // !CUDA_VOLUME_RENDERER_CU
